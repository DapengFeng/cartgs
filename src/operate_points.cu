#include "hip/hip_runtime.h"
/**
 * This file is part of Photo-SLAM
 *
 * Copyright (C) 2023-2024 Longwei Li and Hui Cheng, Sun Yat-sen University.
 * Copyright (C) 2023-2024 Huajian Huang and Sai-Kit Yeung, Hong Kong University
 * of Science and Technology.
 *
 * Photo-SLAM is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * Photo-SLAM is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE. See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with
 * Photo-SLAM. If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <fstream>
#include <iostream>
#include <numeric>

#include "include/operate_points.h"
namespace cg = cooperative_groups;

#include "cuda_rasterizer/operate_points.h"

__global__ void transform_points(const int P,
                                 const float* __restrict__ orig_points,
                                 const float* __restrict__ transformmatrix,
                                 float* __restrict__ trans_points) {
  auto idx = cg::this_grid().thread_rank();
  if (idx >= P) return;

  float3 p_trans = transform_point(idx, orig_points, transformmatrix);
  insert_point_to_pcd(idx, p_trans, trans_points);
}

__global__ void scale_and_transform_points(
    const int P,
    const float scale,
    const float* __restrict__ orig_points,
    const float* __restrict__ orig_rots,
    const float* __restrict__ transformmatrix,
    const bool* __restrict__ mask,
    float* __restrict__ trans_points,
    float* __restrict__ trans_rots) {
  auto idx = cg::this_grid().thread_rank();
  if (idx >= P || !mask[idx]) return;

  float3 p_trans =
      scale_and_transform_point(idx, scale, orig_points, transformmatrix);
  insert_point_to_pcd(idx, p_trans, trans_points);

  float4 rot_trans =
      transfrom_quaternion_using_matrix(idx, orig_rots, transformmatrix);
  insert_rot_to_rots(idx, rot_trans, trans_rots);
}

void transformPoints(torch::Tensor& points, torch::Tensor& transformmatrix) {
  if (points.ndimension() != 2 || points.size(1) != 3) {
    AT_ERROR("points must have dimensions (num_points, 3)");
  }

  const int P = points.size(0);
  torch::Tensor transformed_points = torch::zeros_like(points);

  if (P != 0) {
    transform_points<<<(P + 255) / 256, 256>>>(
        P, points.contiguous().data_ptr<float>(),
        transformmatrix.contiguous().data_ptr<float>(),
        transformed_points.contiguous().data_ptr<float>());

    points = transformed_points;
  }
}

void scaleAndTransformThenMarkVisiblePoints(
    torch::Tensor& points,
    torch::Tensor& rots,
    torch::Tensor& point_not_transformed_mask,
    torch::Tensor& point_unstable_mask,
    torch::Tensor& transformmatrix,
    torch::Tensor& viewmatrix,
    torch::Tensor& projmatrix,
    int& num_transformed,
    const float scale) {
  if (points.ndimension() != 2 || points.size(1) != 3) {
    AT_ERROR("points must have dimensions (num_points, 3)");
  }

  torch::Tensor present = markVisible(points, viewmatrix, projmatrix);

  auto num_points = present.size(0);
  if (point_not_transformed_mask.size(0) != num_points ||
      point_unstable_mask.size(0) != num_points) {
    AT_ERROR("points_mask must have dimensions (num_points)");
  }
  torch::Tensor final_mask =
      torch::logical_and(point_not_transformed_mask, point_unstable_mask);
  final_mask = torch::logical_and(final_mask, present);
  num_transformed += final_mask.sum().item<int>();
  const int P = points.size(0);

  if (P != 0) {
    torch::Tensor transformed_points = torch::zeros_like(points);
    torch::Tensor transformed_rots = torch::zeros_like(rots);

    scale_and_transform_points<<<(P + 255) / 256, 256>>>(
        P, scale, points.contiguous().data_ptr<float>(),
        rots.contiguous().data_ptr<float>(),
        transformmatrix.contiguous().data_ptr<float>(),
        final_mask.contiguous().data_ptr<bool>(),
        transformed_points.contiguous().data_ptr<float>(),
        transformed_rots.contiguous().data_ptr<float>());

    points.index_put_({final_mask}, transformed_points.index({final_mask}));
    rots.index_put_({final_mask}, transformed_rots.index({final_mask}));
    point_not_transformed_mask.index_put_(
        {final_mask},
        torch::full({P}, false, point_not_transformed_mask.options())
            .index({final_mask}));
  }
}
