#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <torch/all.h>

#include <algorithm>
#include <iostream>

namespace cg = cooperative_groups;

#define G_00 0.001028380123898387f
#define G_01 0.0075987582094967365f
#define G_02 0.036000773310661316f
#define G_03 0.10936068743467331f
#define G_04 0.21300552785396576f
#define G_05 0.26601171493530273f
#define G_06 0.21300552785396576f
#define G_07 0.10936068743467331f
#define G_08 0.036000773310661316f
#define G_09 0.0075987582094967365f
#define G_10 0.001028380123898387f

#define G_000 0.0000010576f
#define G_001 0.0000078144f
#define G_002 0.0000370225f
#define G_003 0.0001124644f
#define G_004 0.0002190506f
#define G_005 0.0002735612f
#define G_006 0.0002190506f
#define G_007 0.0001124644f
#define G_008 0.0000370225f
#define G_009 0.0000078144f
#define G_010 0.0000010576f
#define G_011 0.0000078144f
#define G_012 0.0000577411f
#define G_013 0.0002735612f
#define G_014 0.0008310054f
#define G_015 0.0016185775f
#define G_016 0.0020213588f
#define G_017 0.0016185775f
#define G_018 0.0008310054f
#define G_019 0.0002735612f
#define G_020 0.0000577411f
#define G_021 0.0000078144f
#define G_022 0.0000370225f
#define G_023 0.0002735612f
#define G_024 0.0012960557f
#define G_025 0.0039370693f
#define G_026 0.0076683639f
#define G_027 0.0095766271f
#define G_028 0.0076683639f
#define G_029 0.0039370693f
#define G_030 0.0012960557f
#define G_031 0.0002735612f
#define G_032 0.0000370225f
#define G_033 0.0001124644f
#define G_034 0.0008310054f
#define G_035 0.0039370693f
#define G_036 0.0119597595f
#define G_037 0.0232944302f
#define G_038 0.0290912241f
#define G_039 0.0232944302f
#define G_040 0.0119597595f
#define G_041 0.0039370693f
#define G_042 0.0008310054f
#define G_043 0.0001124644f
#define G_044 0.0002190506f
#define G_045 0.0016185775f
#define G_046 0.0076683639f
#define G_047 0.0232944302f
#define G_048 0.0453713536f
#define G_049 0.0566619672f
#define G_050 0.0453713536f
#define G_051 0.0232944302f
#define G_052 0.0076683639f
#define G_053 0.0016185775f
#define G_054 0.0002190506f
#define G_055 0.0002735612f
#define G_056 0.0020213588f
#define G_057 0.0095766271f
#define G_058 0.0290912241f
#define G_059 0.0566619672f
#define G_060 0.0707622319f
#define G_061 0.0566619672f
#define G_062 0.0290912241f
#define G_063 0.0095766271f
#define G_064 0.0020213588f
#define G_065 0.0002735612f
#define G_066 0.0002190506f
#define G_067 0.0016185775f
#define G_068 0.0076683639f
#define G_069 0.0232944302f
#define G_070 0.0453713536f
#define G_071 0.0566619672f
#define G_072 0.0453713536f
#define G_073 0.0232944302f
#define G_074 0.0076683639f
#define G_075 0.0016185775f
#define G_076 0.0002190506f
#define G_077 0.0001124644f
#define G_078 0.0008310054f
#define G_079 0.0039370693f
#define G_080 0.0119597595f
#define G_081 0.0232944302f
#define G_082 0.0290912241f
#define G_083 0.0232944302f
#define G_084 0.0119597595f
#define G_085 0.0039370693f
#define G_086 0.0008310054f
#define G_087 0.0001124644f
#define G_088 0.0000370225f
#define G_089 0.0002735612f
#define G_090 0.0012960557f
#define G_091 0.0039370693f
#define G_092 0.0076683639f
#define G_093 0.0095766271f
#define G_094 0.0076683639f
#define G_095 0.0039370693f
#define G_096 0.0012960557f
#define G_097 0.0002735612f
#define G_098 0.0000370225f
#define G_099 0.0000078144f
#define G_100 0.0000577411f
#define G_101 0.0002735612f
#define G_102 0.0008310054f
#define G_103 0.0016185775f
#define G_104 0.0020213588f
#define G_105 0.0016185775f
#define G_106 0.0008310054f
#define G_107 0.0002735612f
#define G_108 0.0000577411f
#define G_109 0.0000078144f
#define G_110 0.0000010576f
#define G_111 0.0000078144f
#define G_112 0.0000370225f
#define G_113 0.0001124644f
#define G_114 0.0002190506f
#define G_115 0.0002735612f
#define G_116 0.0002190506f
#define G_117 0.0001124644f
#define G_118 0.0000370225f
#define G_119 0.0000078144f
#define G_120 0.0000010576f

#define BX 32
#define BY 32
#define BLOCK_DIM 16

template <int C>
__device__ float get_pix_value(const float* img,
                               const int c,
                               const int y,
                               const int x,
                               const int H,
                               const int W) {
  if (x >= W || y >= H || x < 0 || y < 0) {
    return 0.0f;
  } else {
    return img[c * H * W + y * W + x];
  }
}

/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.numIterations1 (OCT 1995), consisting of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */
template <int C>
__global__ void transposeCUDA(float* odata,
                              float* idata,
                              int width,
                              int height) {
  __shared__ float block[BLOCK_DIM][BLOCK_DIM + 1];
  const int num_pix = width * height;

  for (int c = 0; c < C; ++c) {
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if ((xIndex < width) && (yIndex < height)) {
      unsigned int index_in = yIndex * width + xIndex;
      block[threadIdx.y][threadIdx.x] = idata[num_pix * c + index_in];
    }

    __syncthreads();

    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if ((xIndex < height) && (yIndex < width)) {
      unsigned int index_out = yIndex * height + xIndex;
      odata[num_pix * c + index_out] = block[threadIdx.x][threadIdx.y];
    }
    __syncthreads();
  }
}

template <int C>
__global__ void separableConvCUDA(const float* __restrict__ input,
                                  float* __restrict__ output,
                                  const int H,
                                  const int W) {
  auto block = cg::this_thread_block();
  const int pix_y =
      block.group_index().y * block.dim_threads().y + block.thread_index().y;
  const int pix_x =
      block.group_index().x * block.dim_threads().x + block.thread_index().x;
  const int pix_id = pix_y * W + pix_x;
  const int num_pix = H * W;

  __shared__ float pixels[BY][BX + 10];
  const int start_y = block.group_index().y * block.dim_threads().y;
  const int start_x = block.group_index().x * block.dim_threads().x;

  const int cnt = BY * (BX + 10);
  const int num_blocks = (cnt + BX * BY - 1) / (BX * BY);

  for (int i = 0; i < C; ++i) {
    for (int b = 0; b < num_blocks; ++b) {
      int tid = b * (BX * BY) + block.thread_rank();
      if (tid < cnt) {
        int local_y = tid / (BX + 10);
        int local_x = tid % (BX + 10);
        int y = start_y + local_y;
        int x = start_x + local_x;
        pixels[local_y][local_x] = get_pix_value<C>(input, i, y, x - 5, H, W);
      }
    }
    block.sync();

    if (pix_x < W && pix_y < H) {
      int local_y = block.thread_index().y;
      int local_x = block.thread_index().x + 5;
      float val = 0.0f;
      val += G_00 * pixels[local_y][local_x - 5];
      val += G_01 * pixels[local_y][local_x - 4];
      val += G_02 * pixels[local_y][local_x - 3];
      val += G_03 * pixels[local_y][local_x - 2];
      val += G_04 * pixels[local_y][local_x - 1];
      val += G_05 * pixels[local_y][local_x];
      val += G_06 * pixels[local_y][local_x + 1];
      val += G_07 * pixels[local_y][local_x + 2];
      val += G_08 * pixels[local_y][local_x + 3];
      val += G_09 * pixels[local_y][local_x + 4];
      val += G_10 * pixels[local_y][local_x + 5];
      output[i * num_pix + pix_id] = val;
    }
    block.sync();
  }
}

template <int C>
__global__ void convCUDA(const float* __restrict__ input,
                         float* __restrict__ output,
                         const int H,
                         const int W) {
  auto block = cg::this_thread_block();
  const int pix_y =
      block.group_index().y * block.dim_threads().y + block.thread_index().y;
  const int pix_x =
      block.group_index().x * block.dim_threads().x + block.thread_index().x;
  const int pix_id = pix_y * W + pix_x;
  const int num_pix = H * W;

  __shared__ float pixels[BY + 10][BX + 10];
  const int start_y = block.group_index().y * block.dim_threads().y;
  const int start_x = block.group_index().x * block.dim_threads().x;

  const int cnt = (BY + 10) * (BX + 10);
  const int num_blocks = (cnt + BX * BY - 1) / (BX * BY);

  for (int i = 0; i < C; ++i) {
    for (int b = 0; b < num_blocks; ++b) {
      int tid = b * (BX * BY) + block.thread_rank();
      if (tid < cnt) {
        int local_y = tid / (BX + 10);
        int local_x = tid % (BX + 10);
        int y = start_y + local_y;
        int x = start_x + local_x;
        pixels[local_y][local_x] =
            get_pix_value<C>(input, i, y - 5, x - 5, H, W);
      }
    }
    block.sync();

    if (pix_x < W && pix_y < H) {
      int local_y = block.thread_index().y + 5;
      int local_x = block.thread_index().x + 5;
      float val = 0.0f;

      {
        val += G_000 * pixels[local_y - 5][local_x - 5];
        val += G_001 * pixels[local_y - 5][local_x - 4];
        val += G_002 * pixels[local_y - 5][local_x - 3];
        val += G_003 * pixels[local_y - 5][local_x - 2];
        val += G_004 * pixels[local_y - 5][local_x - 1];
        val += G_005 * pixels[local_y - 5][local_x];
        val += G_006 * pixels[local_y - 5][local_x + 1];
        val += G_007 * pixels[local_y - 5][local_x + 2];
        val += G_008 * pixels[local_y - 5][local_x + 3];
        val += G_009 * pixels[local_y - 5][local_x + 4];
        val += G_010 * pixels[local_y - 5][local_x + 5];
        val += G_011 * pixels[local_y - 4][local_x - 5];
        val += G_012 * pixels[local_y - 4][local_x - 4];
        val += G_013 * pixels[local_y - 4][local_x - 3];
        val += G_014 * pixels[local_y - 4][local_x - 2];
        val += G_015 * pixels[local_y - 4][local_x - 1];
        val += G_016 * pixels[local_y - 4][local_x];
        val += G_017 * pixels[local_y - 4][local_x + 1];
        val += G_018 * pixels[local_y - 4][local_x + 2];
        val += G_019 * pixels[local_y - 4][local_x + 3];
        val += G_020 * pixels[local_y - 4][local_x + 4];
        val += G_021 * pixels[local_y - 4][local_x + 5];
        val += G_022 * pixels[local_y - 3][local_x - 5];
        val += G_023 * pixels[local_y - 3][local_x - 4];
        val += G_024 * pixels[local_y - 3][local_x - 3];
        val += G_025 * pixels[local_y - 3][local_x - 2];
        val += G_026 * pixels[local_y - 3][local_x - 1];
        val += G_027 * pixels[local_y - 3][local_x];
        val += G_028 * pixels[local_y - 3][local_x + 1];
        val += G_029 * pixels[local_y - 3][local_x + 2];
        val += G_030 * pixels[local_y - 3][local_x + 3];
        val += G_031 * pixels[local_y - 3][local_x + 4];
        val += G_032 * pixels[local_y - 3][local_x + 5];
        val += G_033 * pixels[local_y - 2][local_x - 5];
        val += G_034 * pixels[local_y - 2][local_x - 4];
        val += G_035 * pixels[local_y - 2][local_x - 3];
        val += G_036 * pixels[local_y - 2][local_x - 2];
        val += G_037 * pixels[local_y - 2][local_x - 1];
        val += G_038 * pixels[local_y - 2][local_x];
        val += G_039 * pixels[local_y - 2][local_x + 1];
        val += G_040 * pixels[local_y - 2][local_x + 2];
        val += G_041 * pixels[local_y - 2][local_x + 3];
        val += G_042 * pixels[local_y - 2][local_x + 4];
        val += G_043 * pixels[local_y - 2][local_x + 5];
        val += G_044 * pixels[local_y - 1][local_x - 5];
        val += G_045 * pixels[local_y - 1][local_x - 4];
        val += G_046 * pixels[local_y - 1][local_x - 3];
        val += G_047 * pixels[local_y - 1][local_x - 2];
        val += G_048 * pixels[local_y - 1][local_x - 1];
        val += G_049 * pixels[local_y - 1][local_x];
        val += G_050 * pixels[local_y - 1][local_x + 1];
        val += G_051 * pixels[local_y - 1][local_x + 2];
        val += G_052 * pixels[local_y - 1][local_x + 3];
        val += G_053 * pixels[local_y - 1][local_x + 4];
        val += G_054 * pixels[local_y - 1][local_x + 5];
        val += G_055 * pixels[local_y][local_x - 5];
        val += G_056 * pixels[local_y][local_x - 4];
        val += G_057 * pixels[local_y][local_x - 3];
        val += G_058 * pixels[local_y][local_x - 2];
        val += G_059 * pixels[local_y][local_x - 1];
        val += G_060 * pixels[local_y][local_x];
        val += G_061 * pixels[local_y][local_x + 1];
        val += G_062 * pixels[local_y][local_x + 2];
        val += G_063 * pixels[local_y][local_x + 3];
        val += G_064 * pixels[local_y][local_x + 4];
        val += G_065 * pixels[local_y][local_x + 5];
        val += G_066 * pixels[local_y + 1][local_x - 5];
        val += G_067 * pixels[local_y + 1][local_x - 4];
        val += G_068 * pixels[local_y + 1][local_x - 3];
        val += G_069 * pixels[local_y + 1][local_x - 2];
        val += G_070 * pixels[local_y + 1][local_x - 1];
        val += G_071 * pixels[local_y + 1][local_x];
        val += G_072 * pixels[local_y + 1][local_x + 1];
        val += G_073 * pixels[local_y + 1][local_x + 2];
        val += G_074 * pixels[local_y + 1][local_x + 3];
        val += G_075 * pixels[local_y + 1][local_x + 4];
        val += G_076 * pixels[local_y + 1][local_x + 5];
        val += G_077 * pixels[local_y + 2][local_x - 5];
        val += G_078 * pixels[local_y + 2][local_x - 4];
        val += G_079 * pixels[local_y + 2][local_x - 3];
        val += G_080 * pixels[local_y + 2][local_x - 2];
        val += G_081 * pixels[local_y + 2][local_x - 1];
        val += G_082 * pixels[local_y + 2][local_x];
        val += G_083 * pixels[local_y + 2][local_x + 1];
        val += G_084 * pixels[local_y + 2][local_x + 2];
        val += G_085 * pixels[local_y + 2][local_x + 3];
        val += G_086 * pixels[local_y + 2][local_x + 4];
        val += G_087 * pixels[local_y + 2][local_x + 5];
        val += G_088 * pixels[local_y + 3][local_x - 5];
        val += G_089 * pixels[local_y + 3][local_x - 4];
        val += G_090 * pixels[local_y + 3][local_x - 3];
        val += G_091 * pixels[local_y + 3][local_x - 2];
        val += G_092 * pixels[local_y + 3][local_x - 1];
        val += G_093 * pixels[local_y + 3][local_x];
        val += G_094 * pixels[local_y + 3][local_x + 1];
        val += G_095 * pixels[local_y + 3][local_x + 2];
        val += G_096 * pixels[local_y + 3][local_x + 3];
        val += G_097 * pixels[local_y + 3][local_x + 4];
        val += G_098 * pixels[local_y + 3][local_x + 5];
        val += G_099 * pixels[local_y + 4][local_x - 5];
        val += G_100 * pixels[local_y + 4][local_x - 4];
        val += G_101 * pixels[local_y + 4][local_x - 3];
        val += G_102 * pixels[local_y + 4][local_x - 2];
        val += G_103 * pixels[local_y + 4][local_x - 1];
        val += G_104 * pixels[local_y + 4][local_x];
        val += G_105 * pixels[local_y + 4][local_x + 1];
        val += G_106 * pixels[local_y + 4][local_x + 2];
        val += G_107 * pixels[local_y + 4][local_x + 3];
        val += G_108 * pixels[local_y + 4][local_x + 4];
        val += G_109 * pixels[local_y + 4][local_x + 5];
        val += G_110 * pixels[local_y + 5][local_x - 5];
        val += G_111 * pixels[local_y + 5][local_x - 4];
        val += G_112 * pixels[local_y + 5][local_x - 3];
        val += G_113 * pixels[local_y + 5][local_x - 2];
        val += G_114 * pixels[local_y + 5][local_x - 1];
        val += G_115 * pixels[local_y + 5][local_x];
        val += G_116 * pixels[local_y + 5][local_x + 1];
        val += G_117 * pixels[local_y + 5][local_x + 2];
        val += G_118 * pixels[local_y + 5][local_x + 3];
        val += G_119 * pixels[local_y + 5][local_x + 4];
        val += G_120 * pixels[local_y + 5][local_x + 5];
      }

      output[i * num_pix + pix_id] = val;
    }
    block.sync();
  }
}

torch::Tensor conv2DForward(torch::Tensor& input) {
  int H = input.size(1);
  int W = input.size(2);
  dim3 grid((W + BX - 1) / BX, (H + BY - 1) / BY, 1);
  dim3 block(BX, BY, 1);

  torch::Tensor aux = torch::zeros({3, H, W}, input.options());
  convCUDA<3><<<grid, block>>>(input.contiguous().data<float>(),
                               aux.contiguous().data<float>(), H, W);
  return aux;

  separableConvCUDA<3><<<grid, block>>>(input.contiguous().data<float>(),
                                        aux.contiguous().data<float>(), H, W);

  // torch::Tensor aux_T = torch::full({3, W, H}, 0, input.options());
  // grid = dim3((W + BLOCK_DIM - 1) / BLOCK_DIM, (H + BLOCK_DIM - 1) /
  // BLOCK_DIM, 1);
  // block = dim3(BLOCK_DIM, BLOCK_DIM, 1);
  // transposeCUDA<3><<<grid, block>>>(
  //     aux_T.contiguous().data<float>(),
  //     aux.contiguous().data<float>(),
  //     W,
  //     H);

  aux = aux.transpose(1, 2);

  std::swap(H, W);

  torch::Tensor output_T = torch::full({3, H, W}, 0, input.options());
  grid = dim3((W + BX - 1) / BX, (H + BY - 1) / BY, 1);
  block = dim3(BX, BY, 1);
  separableConvCUDA<3><<<grid, block>>>(aux.contiguous().data<float>(),
                                        output_T.contiguous().data<float>(), H,
                                        W);

  // torch::Tensor output = torch::full({3, W, H}, 0, input.options());
  // grid = dim3((W + BLOCK_DIM - 1) / BLOCK_DIM, (H + BLOCK_DIM - 1) /
  // BLOCK_DIM, 1);
  // block = dim3(BLOCK_DIM, BLOCK_DIM, 1);
  // transposeCUDA<3><<<grid, block>>>(
  //     output.contiguous().data<float>(),
  //     output_T.contiguous().data<float>(),
  //     W,
  //     H);
  // std::swap(H, W);
  return output_T.transpose(1, 2);
}

__global__ void ssimrestCUDA(int N,
                             float C1,
                             float C2,
                             float* mu1,
                             float* mu2,
                             float* mim,
                             float* mom,
                             float* mu2_sq,
                             float* sigma2_sq,
                             float* ssim_map) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= N) return;

  float mu1_sq = mu1[idx] * mu1[idx];
  float mu1_mu2 = mu1[idx] * mu2[idx];
  float sigma1_sq = mim[idx] - mu1_sq;
  float sigma12 = mom[idx] - mu1_mu2;

  ssim_map[idx] =
      ((2.0f * mu1_mu2 + C1) * (2.0f * sigma12 + C2)) /
      ((mu1_sq + mu2_sq[idx] + C1) * (sigma1_sq + sigma2_sq[idx] + C2));
}

__global__ void ssimrest_backCUDA(int N,
                                  float C1,
                                  float C2,
                                  float* mu1_,
                                  float* mu2_,
                                  float* mim_,
                                  float* mom_,
                                  float* mu2_sq_,
                                  float* sigma2_sq_,
                                  float* dL,
                                  float* dL_dmu1,
                                  float* dL_dmim,
                                  float* dL_dmom) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= N) return;

  float mu1 = mu1_[idx];
  float mu2 = mu2_[idx];
  float mu2_sq = mu2_sq_[idx];
  float mim = mim_[idx];
  float mom = mom_[idx];
  float sigma2_sq = sigma2_sq_[idx];

  float A = (mu1 * mu1 + C1 + mu2_sq);
  float B = (-mu1 * mu1 + C2 + mim + sigma2_sq);
  float C = (C1 + 2 * mu1 * mu2);
  float D = (C2 + 2 * mom - 2 * mu1 * mu2);

  float L = dL[idx];
  dL_dmu1[idx] =
      L * ((2 * mu2 * D) / (A * B) - (2 * mu2 * C) / (A * B) +
           (2 * mu1 * C * D) / (A * B * B) - (2 * mu1 * C * D) / (A * A * B));
  dL_dmim[idx] = L * (-(C * D) / (A * B * B));
  dL_dmom[idx] = L * ((2 * C) / (A * B));
}

__global__ void lol(float* hnk) { hnk[0] = 42; }

torch::Tensor ssimrest(float C1,
                       float C2,
                       torch::Tensor& mu1,
                       torch::Tensor& mu2,
                       torch::Tensor& mim,
                       torch::Tensor& mom,
                       torch::Tensor& mu2_sq,
                       torch::Tensor& sigma2_sq) {
  int N = mu1.size(0) * mu1.size(1) * mu1.size(2);

  torch::Tensor target = torch::zeros_like(mu1).contiguous();
  ssimrestCUDA<<<(N + 255) / 256, 256>>>(
      N, C1, C2, mu1.contiguous().data<float>(), mu2.contiguous().data<float>(),
      mim.contiguous().data<float>(), mom.contiguous().data<float>(),
      mu2_sq.contiguous().data<float>(), sigma2_sq.contiguous().data<float>(),
      target.contiguous().data<float>());
  return target;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> ssimrest_back(
    float C1,
    float C2,
    torch::Tensor& mu1,
    torch::Tensor& mu2,
    torch::Tensor& mim,
    torch::Tensor& mom,
    torch::Tensor& mu2_sq,
    torch::Tensor& sigma2_sq,
    torch::Tensor& dL) {
  int N = mu1.size(0) * mu1.size(1) * mu1.size(2);

  torch::Tensor dL_dmu1 = torch::zeros_like(mu1).contiguous();
  torch::Tensor dL_dmim = torch::zeros_like(mu1).contiguous();
  torch::Tensor dL_dmom = torch::zeros_like(mu1).contiguous();
  ssimrest_backCUDA<<<(N + 255) / 256, 256>>>(
      N, C1, C2, mu1.contiguous().data<float>(), mu2.contiguous().data<float>(),
      mim.contiguous().data<float>(), mom.contiguous().data<float>(),
      mu2_sq.contiguous().data<float>(), sigma2_sq.contiguous().data<float>(),
      dL.contiguous().data<float>(), dL_dmu1.contiguous().data<float>(),
      dL_dmim.contiguous().data<float>(), dL_dmom.contiguous().data<float>());
  return std::make_tuple(dL_dmu1, dL_dmim, dL_dmom);
}

template <int C>
__device__ void load_into_shared(float pixels[BY + 10][BX + 10],
                                 float* input1,
                                 float* input2,
                                 int H,
                                 int W,
                                 int i,
                                 int subtract = 0) {
  auto block = cg::this_thread_block();
  const int start_y = block.group_index().y * (BY - subtract) - subtract / 2;
  const int start_x = block.group_index().x * (BX - subtract) - subtract / 2;

  const int cnt = (BY + 10) * (BX + 10);
  const int num_blocks = (cnt + BX * BY - 1) / (BX * BY);
  for (int b = 0; b < num_blocks; ++b) {
    int tid = b * (BX * BY) + block.thread_rank();
    if (tid < cnt) {
      int local_y = tid / (BX + 10);
      int local_x = tid % (BX + 10);
      int y = start_y + local_y;
      int x = start_x + local_x;
      if (input2 == nullptr) {
        float one = get_pix_value<C>(input1, i, y - 5, x - 5, H, W);
        pixels[local_y][local_x] = one;
      } else {
        float one = get_pix_value<C>(input1, i, y - 5, x - 5, H, W);
        float two = get_pix_value<C>(input2, i, y - 5, x - 5, H, W);
        pixels[local_y][local_x] = one * two;
      }
    }
  }
}

__device__ void write_to_shared(float pixels[BY + 10][BX + 10], float val) {
  auto block = cg::this_thread_block();

  // flush with 0s
  const int cnt = (BY + 10) * (BX + 10);
  const int num_blocks = (cnt + BX * BY - 1) / (BX * BY);
  for (int b = 0; b < num_blocks; ++b) {
    int tid = b * (BX * BY) + block.thread_rank();
    if (tid < cnt) {
      int local_y = tid / (BX + 10);
      int local_x = tid % (BX + 10);
      pixels[local_y][local_x] = 0.0f;
    }
  }
  block.sync();

  // write the values in the central BXxBY zone
  pixels[block.thread_index().y + 5][block.thread_index().x + 5] = val;
}

__device__ void multiply_shared_mem(float pix1[BY + 10][BX + 10],
                                    float pix2[BY + 10][BX + 10]) {
  auto block = cg::this_thread_block();
  const int cnt = (BY + 10) * (BX + 10);
  const int num_blocks = (cnt + BX * BY - 1) / (BX * BY);
  for (int b = 0; b < num_blocks; ++b) {
    int tid = b * (BX * BY) + block.thread_rank();
    if (tid < cnt) {
      int local_y = tid / (BX + 10);
      int local_x = tid % (BX + 10);
      float one = pix1[local_y][local_x];
      float two = pix2[local_y][local_x];
      pix1[local_y][local_x] = one * two;
    }
  }
}

__device__ inline float do_sq(float val) { return val * val; }

__device__ float do_conv(float pixels[BY + 10][BX + 10],
                         int H,
                         int W,
                         bool sq = false) {
  auto block = cg::this_thread_block();
  int local_y = block.thread_index().y + 5;
  int local_x = block.thread_index().x + 5;
  float val = 0.0f;

  if (sq) {
    val += G_000 * do_sq(pixels[local_y - 5][local_x - 5]);
    val += G_001 * do_sq(pixels[local_y - 5][local_x - 4]);
    val += G_002 * do_sq(pixels[local_y - 5][local_x - 3]);
    val += G_003 * do_sq(pixels[local_y - 5][local_x - 2]);
    val += G_004 * do_sq(pixels[local_y - 5][local_x - 1]);
    val += G_005 * do_sq(pixels[local_y - 5][local_x]);
    val += G_006 * do_sq(pixels[local_y - 5][local_x + 1]);
    val += G_007 * do_sq(pixels[local_y - 5][local_x + 2]);
    val += G_008 * do_sq(pixels[local_y - 5][local_x + 3]);
    val += G_009 * do_sq(pixels[local_y - 5][local_x + 4]);
    val += G_010 * do_sq(pixels[local_y - 5][local_x + 5]);
    val += G_011 * do_sq(pixels[local_y - 4][local_x - 5]);
    val += G_012 * do_sq(pixels[local_y - 4][local_x - 4]);
    val += G_013 * do_sq(pixels[local_y - 4][local_x - 3]);
    val += G_014 * do_sq(pixels[local_y - 4][local_x - 2]);
    val += G_015 * do_sq(pixels[local_y - 4][local_x - 1]);
    val += G_016 * do_sq(pixels[local_y - 4][local_x]);
    val += G_017 * do_sq(pixels[local_y - 4][local_x + 1]);
    val += G_018 * do_sq(pixels[local_y - 4][local_x + 2]);
    val += G_019 * do_sq(pixels[local_y - 4][local_x + 3]);
    val += G_020 * do_sq(pixels[local_y - 4][local_x + 4]);
    val += G_021 * do_sq(pixels[local_y - 4][local_x + 5]);
    val += G_022 * do_sq(pixels[local_y - 3][local_x - 5]);
    val += G_023 * do_sq(pixels[local_y - 3][local_x - 4]);
    val += G_024 * do_sq(pixels[local_y - 3][local_x - 3]);
    val += G_025 * do_sq(pixels[local_y - 3][local_x - 2]);
    val += G_026 * do_sq(pixels[local_y - 3][local_x - 1]);
    val += G_027 * do_sq(pixels[local_y - 3][local_x]);
    val += G_028 * do_sq(pixels[local_y - 3][local_x + 1]);
    val += G_029 * do_sq(pixels[local_y - 3][local_x + 2]);
    val += G_030 * do_sq(pixels[local_y - 3][local_x + 3]);
    val += G_031 * do_sq(pixels[local_y - 3][local_x + 4]);
    val += G_032 * do_sq(pixels[local_y - 3][local_x + 5]);
    val += G_033 * do_sq(pixels[local_y - 2][local_x - 5]);
    val += G_034 * do_sq(pixels[local_y - 2][local_x - 4]);
    val += G_035 * do_sq(pixels[local_y - 2][local_x - 3]);
    val += G_036 * do_sq(pixels[local_y - 2][local_x - 2]);
    val += G_037 * do_sq(pixels[local_y - 2][local_x - 1]);
    val += G_038 * do_sq(pixels[local_y - 2][local_x]);
    val += G_039 * do_sq(pixels[local_y - 2][local_x + 1]);
    val += G_040 * do_sq(pixels[local_y - 2][local_x + 2]);
    val += G_041 * do_sq(pixels[local_y - 2][local_x + 3]);
    val += G_042 * do_sq(pixels[local_y - 2][local_x + 4]);
    val += G_043 * do_sq(pixels[local_y - 2][local_x + 5]);
    val += G_044 * do_sq(pixels[local_y - 1][local_x - 5]);
    val += G_045 * do_sq(pixels[local_y - 1][local_x - 4]);
    val += G_046 * do_sq(pixels[local_y - 1][local_x - 3]);
    val += G_047 * do_sq(pixels[local_y - 1][local_x - 2]);
    val += G_048 * do_sq(pixels[local_y - 1][local_x - 1]);
    val += G_049 * do_sq(pixels[local_y - 1][local_x]);
    val += G_050 * do_sq(pixels[local_y - 1][local_x + 1]);
    val += G_051 * do_sq(pixels[local_y - 1][local_x + 2]);
    val += G_052 * do_sq(pixels[local_y - 1][local_x + 3]);
    val += G_053 * do_sq(pixels[local_y - 1][local_x + 4]);
    val += G_054 * do_sq(pixels[local_y - 1][local_x + 5]);
    val += G_055 * do_sq(pixels[local_y][local_x - 5]);
    val += G_056 * do_sq(pixels[local_y][local_x - 4]);
    val += G_057 * do_sq(pixels[local_y][local_x - 3]);
    val += G_058 * do_sq(pixels[local_y][local_x - 2]);
    val += G_059 * do_sq(pixels[local_y][local_x - 1]);
    val += G_060 * do_sq(pixels[local_y][local_x]);
    val += G_061 * do_sq(pixels[local_y][local_x + 1]);
    val += G_062 * do_sq(pixels[local_y][local_x + 2]);
    val += G_063 * do_sq(pixels[local_y][local_x + 3]);
    val += G_064 * do_sq(pixels[local_y][local_x + 4]);
    val += G_065 * do_sq(pixels[local_y][local_x + 5]);
    val += G_066 * do_sq(pixels[local_y + 1][local_x - 5]);
    val += G_067 * do_sq(pixels[local_y + 1][local_x - 4]);
    val += G_068 * do_sq(pixels[local_y + 1][local_x - 3]);
    val += G_069 * do_sq(pixels[local_y + 1][local_x - 2]);
    val += G_070 * do_sq(pixels[local_y + 1][local_x - 1]);
    val += G_071 * do_sq(pixels[local_y + 1][local_x]);
    val += G_072 * do_sq(pixels[local_y + 1][local_x + 1]);
    val += G_073 * do_sq(pixels[local_y + 1][local_x + 2]);
    val += G_074 * do_sq(pixels[local_y + 1][local_x + 3]);
    val += G_075 * do_sq(pixels[local_y + 1][local_x + 4]);
    val += G_076 * do_sq(pixels[local_y + 1][local_x + 5]);
    val += G_077 * do_sq(pixels[local_y + 2][local_x - 5]);
    val += G_078 * do_sq(pixels[local_y + 2][local_x - 4]);
    val += G_079 * do_sq(pixels[local_y + 2][local_x - 3]);
    val += G_080 * do_sq(pixels[local_y + 2][local_x - 2]);
    val += G_081 * do_sq(pixels[local_y + 2][local_x - 1]);
    val += G_082 * do_sq(pixels[local_y + 2][local_x]);
    val += G_083 * do_sq(pixels[local_y + 2][local_x + 1]);
    val += G_084 * do_sq(pixels[local_y + 2][local_x + 2]);
    val += G_085 * do_sq(pixels[local_y + 2][local_x + 3]);
    val += G_086 * do_sq(pixels[local_y + 2][local_x + 4]);
    val += G_087 * do_sq(pixels[local_y + 2][local_x + 5]);
    val += G_088 * do_sq(pixels[local_y + 3][local_x - 5]);
    val += G_089 * do_sq(pixels[local_y + 3][local_x - 4]);
    val += G_090 * do_sq(pixels[local_y + 3][local_x - 3]);
    val += G_091 * do_sq(pixels[local_y + 3][local_x - 2]);
    val += G_092 * do_sq(pixels[local_y + 3][local_x - 1]);
    val += G_093 * do_sq(pixels[local_y + 3][local_x]);
    val += G_094 * do_sq(pixels[local_y + 3][local_x + 1]);
    val += G_095 * do_sq(pixels[local_y + 3][local_x + 2]);
    val += G_096 * do_sq(pixels[local_y + 3][local_x + 3]);
    val += G_097 * do_sq(pixels[local_y + 3][local_x + 4]);
    val += G_098 * do_sq(pixels[local_y + 3][local_x + 5]);
    val += G_099 * do_sq(pixels[local_y + 4][local_x - 5]);
    val += G_100 * do_sq(pixels[local_y + 4][local_x - 4]);
    val += G_101 * do_sq(pixels[local_y + 4][local_x - 3]);
    val += G_102 * do_sq(pixels[local_y + 4][local_x - 2]);
    val += G_103 * do_sq(pixels[local_y + 4][local_x - 1]);
    val += G_104 * do_sq(pixels[local_y + 4][local_x]);
    val += G_105 * do_sq(pixels[local_y + 4][local_x + 1]);
    val += G_106 * do_sq(pixels[local_y + 4][local_x + 2]);
    val += G_107 * do_sq(pixels[local_y + 4][local_x + 3]);
    val += G_108 * do_sq(pixels[local_y + 4][local_x + 4]);
    val += G_109 * do_sq(pixels[local_y + 4][local_x + 5]);
    val += G_110 * do_sq(pixels[local_y + 5][local_x - 5]);
    val += G_111 * do_sq(pixels[local_y + 5][local_x - 4]);
    val += G_112 * do_sq(pixels[local_y + 5][local_x - 3]);
    val += G_113 * do_sq(pixels[local_y + 5][local_x - 2]);
    val += G_114 * do_sq(pixels[local_y + 5][local_x - 1]);
    val += G_115 * do_sq(pixels[local_y + 5][local_x]);
    val += G_116 * do_sq(pixels[local_y + 5][local_x + 1]);
    val += G_117 * do_sq(pixels[local_y + 5][local_x + 2]);
    val += G_118 * do_sq(pixels[local_y + 5][local_x + 3]);
    val += G_119 * do_sq(pixels[local_y + 5][local_x + 4]);
    val += G_120 * do_sq(pixels[local_y + 5][local_x + 5]);
  } else {
    val += G_000 * pixels[local_y - 5][local_x - 5];
    val += G_001 * pixels[local_y - 5][local_x - 4];
    val += G_002 * pixels[local_y - 5][local_x - 3];
    val += G_003 * pixels[local_y - 5][local_x - 2];
    val += G_004 * pixels[local_y - 5][local_x - 1];
    val += G_005 * pixels[local_y - 5][local_x];
    val += G_006 * pixels[local_y - 5][local_x + 1];
    val += G_007 * pixels[local_y - 5][local_x + 2];
    val += G_008 * pixels[local_y - 5][local_x + 3];
    val += G_009 * pixels[local_y - 5][local_x + 4];
    val += G_010 * pixels[local_y - 5][local_x + 5];
    val += G_011 * pixels[local_y - 4][local_x - 5];
    val += G_012 * pixels[local_y - 4][local_x - 4];
    val += G_013 * pixels[local_y - 4][local_x - 3];
    val += G_014 * pixels[local_y - 4][local_x - 2];
    val += G_015 * pixels[local_y - 4][local_x - 1];
    val += G_016 * pixels[local_y - 4][local_x];
    val += G_017 * pixels[local_y - 4][local_x + 1];
    val += G_018 * pixels[local_y - 4][local_x + 2];
    val += G_019 * pixels[local_y - 4][local_x + 3];
    val += G_020 * pixels[local_y - 4][local_x + 4];
    val += G_021 * pixels[local_y - 4][local_x + 5];
    val += G_022 * pixels[local_y - 3][local_x - 5];
    val += G_023 * pixels[local_y - 3][local_x - 4];
    val += G_024 * pixels[local_y - 3][local_x - 3];
    val += G_025 * pixels[local_y - 3][local_x - 2];
    val += G_026 * pixels[local_y - 3][local_x - 1];
    val += G_027 * pixels[local_y - 3][local_x];
    val += G_028 * pixels[local_y - 3][local_x + 1];
    val += G_029 * pixels[local_y - 3][local_x + 2];
    val += G_030 * pixels[local_y - 3][local_x + 3];
    val += G_031 * pixels[local_y - 3][local_x + 4];
    val += G_032 * pixels[local_y - 3][local_x + 5];
    val += G_033 * pixels[local_y - 2][local_x - 5];
    val += G_034 * pixels[local_y - 2][local_x - 4];
    val += G_035 * pixels[local_y - 2][local_x - 3];
    val += G_036 * pixels[local_y - 2][local_x - 2];
    val += G_037 * pixels[local_y - 2][local_x - 1];
    val += G_038 * pixels[local_y - 2][local_x];
    val += G_039 * pixels[local_y - 2][local_x + 1];
    val += G_040 * pixels[local_y - 2][local_x + 2];
    val += G_041 * pixels[local_y - 2][local_x + 3];
    val += G_042 * pixels[local_y - 2][local_x + 4];
    val += G_043 * pixels[local_y - 2][local_x + 5];
    val += G_044 * pixels[local_y - 1][local_x - 5];
    val += G_045 * pixels[local_y - 1][local_x - 4];
    val += G_046 * pixels[local_y - 1][local_x - 3];
    val += G_047 * pixels[local_y - 1][local_x - 2];
    val += G_048 * pixels[local_y - 1][local_x - 1];
    val += G_049 * pixels[local_y - 1][local_x];
    val += G_050 * pixels[local_y - 1][local_x + 1];
    val += G_051 * pixels[local_y - 1][local_x + 2];
    val += G_052 * pixels[local_y - 1][local_x + 3];
    val += G_053 * pixels[local_y - 1][local_x + 4];
    val += G_054 * pixels[local_y - 1][local_x + 5];
    val += G_055 * pixels[local_y][local_x - 5];
    val += G_056 * pixels[local_y][local_x - 4];
    val += G_057 * pixels[local_y][local_x - 3];
    val += G_058 * pixels[local_y][local_x - 2];
    val += G_059 * pixels[local_y][local_x - 1];
    val += G_060 * pixels[local_y][local_x];
    val += G_061 * pixels[local_y][local_x + 1];
    val += G_062 * pixels[local_y][local_x + 2];
    val += G_063 * pixels[local_y][local_x + 3];
    val += G_064 * pixels[local_y][local_x + 4];
    val += G_065 * pixels[local_y][local_x + 5];
    val += G_066 * pixels[local_y + 1][local_x - 5];
    val += G_067 * pixels[local_y + 1][local_x - 4];
    val += G_068 * pixels[local_y + 1][local_x - 3];
    val += G_069 * pixels[local_y + 1][local_x - 2];
    val += G_070 * pixels[local_y + 1][local_x - 1];
    val += G_071 * pixels[local_y + 1][local_x];
    val += G_072 * pixels[local_y + 1][local_x + 1];
    val += G_073 * pixels[local_y + 1][local_x + 2];
    val += G_074 * pixels[local_y + 1][local_x + 3];
    val += G_075 * pixels[local_y + 1][local_x + 4];
    val += G_076 * pixels[local_y + 1][local_x + 5];
    val += G_077 * pixels[local_y + 2][local_x - 5];
    val += G_078 * pixels[local_y + 2][local_x - 4];
    val += G_079 * pixels[local_y + 2][local_x - 3];
    val += G_080 * pixels[local_y + 2][local_x - 2];
    val += G_081 * pixels[local_y + 2][local_x - 1];
    val += G_082 * pixels[local_y + 2][local_x];
    val += G_083 * pixels[local_y + 2][local_x + 1];
    val += G_084 * pixels[local_y + 2][local_x + 2];
    val += G_085 * pixels[local_y + 2][local_x + 3];
    val += G_086 * pixels[local_y + 2][local_x + 4];
    val += G_087 * pixels[local_y + 2][local_x + 5];
    val += G_088 * pixels[local_y + 3][local_x - 5];
    val += G_089 * pixels[local_y + 3][local_x - 4];
    val += G_090 * pixels[local_y + 3][local_x - 3];
    val += G_091 * pixels[local_y + 3][local_x - 2];
    val += G_092 * pixels[local_y + 3][local_x - 1];
    val += G_093 * pixels[local_y + 3][local_x];
    val += G_094 * pixels[local_y + 3][local_x + 1];
    val += G_095 * pixels[local_y + 3][local_x + 2];
    val += G_096 * pixels[local_y + 3][local_x + 3];
    val += G_097 * pixels[local_y + 3][local_x + 4];
    val += G_098 * pixels[local_y + 3][local_x + 5];
    val += G_099 * pixels[local_y + 4][local_x - 5];
    val += G_100 * pixels[local_y + 4][local_x - 4];
    val += G_101 * pixels[local_y + 4][local_x - 3];
    val += G_102 * pixels[local_y + 4][local_x - 2];
    val += G_103 * pixels[local_y + 4][local_x - 1];
    val += G_104 * pixels[local_y + 4][local_x];
    val += G_105 * pixels[local_y + 4][local_x + 1];
    val += G_106 * pixels[local_y + 4][local_x + 2];
    val += G_107 * pixels[local_y + 4][local_x + 3];
    val += G_108 * pixels[local_y + 4][local_x + 4];
    val += G_109 * pixels[local_y + 4][local_x + 5];
    val += G_110 * pixels[local_y + 5][local_x - 5];
    val += G_111 * pixels[local_y + 5][local_x - 4];
    val += G_112 * pixels[local_y + 5][local_x - 3];
    val += G_113 * pixels[local_y + 5][local_x - 2];
    val += G_114 * pixels[local_y + 5][local_x - 1];
    val += G_115 * pixels[local_y + 5][local_x];
    val += G_116 * pixels[local_y + 5][local_x + 1];
    val += G_117 * pixels[local_y + 5][local_x + 2];
    val += G_118 * pixels[local_y + 5][local_x + 3];
    val += G_119 * pixels[local_y + 5][local_x + 4];
    val += G_120 * pixels[local_y + 5][local_x + 5];
  }
  return val;
}

template <int CH>
__global__ void fusedssimCUDA(int H,
                              int W,
                              float C1,
                              float C2,
                              float* img1,
                              float* img2,
                              float* ssim_map) {
  auto block = cg::this_thread_block();
  const int pix_y = block.group_index().y * BY + block.thread_index().y;
  const int pix_x = block.group_index().x * BX + block.thread_index().x;
  const int pix_id = pix_y * W + pix_x;
  const int num_pix = H * W;

  // stats for ssim
  float mu1 = 0.0f;
  float mu2 = 0.0f;
  float sigma1_sq = 0.0f;
  float sigma2_sq = 0.0f;
  float sigma12 = 0.0f;

  // shared memory that will be used to load pixels temporarily
  __shared__ float buf1[BY + 10][BX + 10];
  __shared__ float buf2[BY + 10][BX + 10];

  // mu1 <- Conv(img1)
  // sigma1_sq = Conv(img1 * img1) - mu1_sq
  for (int i = 0; i < CH; ++i) {
    // load into shared
    load_into_shared<CH>(buf1, img1, nullptr, H, W, i);
    block.sync();
    // conv
    mu1 = do_conv(buf1, H, W);
    sigma1_sq = do_conv(buf1, H, W, true) - mu1 * mu1;
    block.sync();

    // mu2 <- Conv(img2)
    // sigma2_sq = Conv(img2 * img2) - mu2_sq
    // load into shared
    load_into_shared<CH>(buf2, img2, nullptr, H, W, i);
    block.sync();
    // conv
    mu2 = do_conv(buf2, H, W);
    sigma2_sq = do_conv(buf2, H, W, true) - mu2 * mu2;
    block.sync();

    // sigma12 = Conv(img1 * img2) - mu1_mu2
    // load into shared
    multiply_shared_mem(buf1, buf2);
    block.sync();
    // conv
    sigma12 = do_conv(buf1, H, W) - mu1 * mu2;
    block.sync();

    float mu1_sq = mu1 * mu1;
    float mu2_sq = mu2 * mu2;
    float mu1_mu2 = mu1 * mu2;
    float C = (2.0f * mu1_mu2 + C1);
    float D = (2.0f * sigma12 + C2);
    float A = (mu1_sq + mu2_sq + C1);
    float B = (sigma1_sq + sigma2_sq + C2);
    float m = (C * D) / (A * B);
    if (pix_x < W && pix_y < H) {
      ssim_map[i * num_pix + pix_id] = m;
    }
  }
}

__device__ bool in_inner_window() {
  auto block = cg::this_thread_block();
  return 5 <= block.thread_index().y && block.thread_index().y < BY - 5 &&
         5 <= block.thread_index().x && block.thread_index().x < BX - 5;
}

template <int CH>
__global__ void fusedssim_backwardCUDA(int H,
                                       int W,
                                       float C1,
                                       float C2,
                                       float* img1,
                                       float* img2,
                                       float* dL_dmap,
                                       float* dL_dimg1) {
  auto block = cg::this_thread_block();
  const int pix_y =
      block.group_index().y * (BY - 10) + block.thread_index().y - 5;
  const int pix_x =
      block.group_index().x * (BX - 10) + block.thread_index().x - 5;
  const int pix_id = pix_y * W + pix_x;
  const int num_pix = H * W;

  // stats for ssim
  float mu1 = 0.0f;
  float mu2 = 0.0f;
  float sigma1_sq = 0.0f;
  float sigma2_sq = 0.0f;
  float sigma12 = 0.0f;

  // shared memory that will be used to load pixels temporarily
  __shared__ float buf1[BY + 10][BX + 10];
  __shared__ float buf2[BY + 10][BX + 10];

  // mu1 <- Conv(img1)
  // sigma1_sq = Conv(img1 * img1) - mu1_sq
  for (int i = 0; i < CH; ++i) {
    // load into shared
    load_into_shared<CH>(buf1, img1, nullptr, H, W, i, 10);
    block.sync();
    // conv
    mu1 = do_conv(buf1, H, W);
    sigma1_sq = do_conv(buf1, H, W, true) - mu1 * mu1;
    block.sync();

    // mu2 <- Conv(img2)
    // sigma2_sq = Conv(img2 * img2) - mu2_sq
    // load into shared
    load_into_shared<CH>(buf2, img2, nullptr, H, W, i, 10);
    block.sync();
    // conv
    mu2 = do_conv(buf2, H, W);
    sigma2_sq = do_conv(buf2, H, W, true) - mu2 * mu2;
    block.sync();

    // sigma12 = Conv(img1 * img2) - mu1_mu2
    // load into shared
    multiply_shared_mem(buf2, buf1);
    block.sync();
    // conv
    sigma12 = do_conv(buf2, H, W) - mu1 * mu2;
    block.sync();

    float mu1_sq = mu1 * mu1;
    float mu2_sq = mu2 * mu2;
    float mu1_mu2 = mu1 * mu2;
    float C = (2.0f * mu1_mu2 + C1);
    float D = (2.0f * sigma12 + C2);
    float A = (mu1_sq + mu2_sq + C1);
    float B = (sigma1_sq + sigma2_sq + C2);
    float m = (C * D) / (A * B);
    // if (in_inner_window() && pix_x < W && pix_y < H) {
    //     ssim_map[i * num_pix + pix_id] = m;
    //     MU1[i * num_pix + pix_id] = mu1;
    //     MU2[i * num_pix + pix_id] = mu2;
    //     SIGMA1_SQ[i * num_pix + pix_id] = sigma1_sq;
    //     SIGMA2_SQ[i * num_pix + pix_id] = sigma2_sq;
    //     SIGMA12[i * num_pix + pix_id] = sigma12;
    // }

    float dL_dm = 0.0f;
    if (in_inner_window() && pix_x < W && pix_y < H)
      dL_dm = dL_dmap[i * num_pix + pix_id];
    float dL_dmu1 =
        dL_dm * ((mu2 * 2.0f * D) / (A * B) - (mu2 * 2.0f * C) / (A * B) -
                 (mu1 * 2.0f * C * D) / (A * A * B) +
                 (mu1 * 2.0f * C * D) / (A * B * B));
    float dL_dsigma1_sq = dL_dm * ((-C * D) / (A * B * B));
    float dL_dsigma12 = dL_dm * ((2 * C) / (A * B));

    float dL_dpix = 0.0f;
    float tmp = 0.0f;

    // gradient from mu1
    write_to_shared(buf2, dL_dmu1);
    block.sync();
    tmp = do_conv(buf2, H, W);
    block.sync();
    dL_dpix += tmp;

    // gradient from sigma1_sq
    write_to_shared(buf2, dL_dsigma1_sq);
    block.sync();
    // tmp = get_pix_value<CH>(img1, i, pix_y, pix_x, H, W);
    tmp = buf1[block.thread_index().y + 5][block.thread_index().x + 5];
    tmp *= 2.0f * do_conv(buf2, H, W);
    block.sync();
    dL_dpix += tmp;
    // write_to_shared(buf2, dL_dsigma1_sq * mu1);
    // block.sync();
    // tmp = -2.0f * do_conv(buf2, H, W);
    // block.sync();
    // dL_dpix += tmp;

    // gradient from sigma12
    write_to_shared(buf2, dL_dsigma12);
    block.sync();
    tmp = get_pix_value<CH>(img2, i, pix_y, pix_x, H, W);
    tmp *= do_conv(buf2, H, W);
    block.sync();
    dL_dpix += tmp;
    // write_to_shared(buf2, dL_dsigma12 * mu2);
    // block.sync();
    // tmp = - do_conv(buf2, H, W);
    // block.sync();
    // dL_dpix += tmp;

    if (in_inner_window() && pix_x < W && pix_y < H)
      dL_dimg1[i * num_pix + pix_id] = dL_dpix;
  }
}

torch::Tensor fusedssim(float C1,
                        float C2,
                        const torch::Tensor& img1,
                        const torch::Tensor& img2) {
  int H = img1.size(1);
  int W = img1.size(2);
  dim3 grid((W + BX - 1) / BX, (H + BY - 1) / BY, 1);
  dim3 block(BX, BY, 1);
  // dim3 grid((W + (BX - 10) - 1) / (BX - 10), (H + (BY - 10) - 1) / (BY - 10),
  // 1); dim3 block(BX, BY, 1);

  torch::Tensor target = torch::zeros_like(img1).contiguous();
  fusedssimCUDA<3><<<grid, block>>>(
      H, W, C1, C2, img1.contiguous().data<float>(),
      img2.contiguous().data<float>(), target.contiguous().data<float>());

  return target;
}

torch::Tensor fusedssim_backward(float C1,
                                 float C2,
                                 const torch::Tensor& img1,
                                 const torch::Tensor& img2,
                                 const torch::Tensor& dL_dmap) {
  int H = img1.size(1);
  int W = img1.size(2);
  dim3 grid((W + (BX - 10) - 1) / (BX - 10), (H + (BY - 10) - 1) / (BY - 10),
            1);
  dim3 block(BX, BY, 1);

  torch::Tensor dL_dimg1 = torch::zeros_like(img1).contiguous();

  fusedssim_backwardCUDA<3><<<grid, block>>>(
      H, W, C1, C2, img1.contiguous().data<float>(),
      img2.contiguous().data<float>(), dL_dmap.contiguous().data<float>(),
      dL_dimg1.contiguous().data<float>());

  return dL_dimg1;
}
