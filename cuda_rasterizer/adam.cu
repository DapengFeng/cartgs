#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2024 {Mallick and Goel} and Kerbl, Bernhard and Vicente
// Carrasco, Francisco and Steinberger, Markus and De La Torre, Fernando

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.

// The Software constitues modifications to the 3D Gaussian Splatting codebase,
// which is licensed according to the text in "LICENSE_ORIGINAL". ONLY
// modifications made by the authors are licensed under the MIT License. To
// facilitate the identification of the modifications licensed under the MIT
// License, a diff file ("changes") highlighting these modifications is
// included.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "adam.h"
#include "auxiliary.h"
namespace cg = cooperative_groups;

// step on a grid of size (N, M)
// N is always number of gaussians
__global__ void adamUpdateCUDA(float* __restrict__ param,
                               const float* __restrict__ param_grad,
                               float* __restrict__ exp_avg,
                               float* __restrict__ exp_avg_sq,
                               const bool* tiles_touched,
                               const float lr,
                               const float b1,
                               const float b2,
                               const float eps,
                               const uint32_t N,
                               const uint32_t M) {
  auto p_idx = cg::this_grid().thread_rank();
  const uint32_t g_idx = p_idx / M;
  if (g_idx >= N) return;
  if (tiles_touched[g_idx]) {
    float Register_param_grad = param_grad[p_idx];
    float Register_exp_avg = exp_avg[p_idx];
    float Register_exp_avg_sq = exp_avg_sq[p_idx];
    Register_exp_avg =
        b1 * Register_exp_avg + (1.0f - b1) * Register_param_grad;
    Register_exp_avg_sq = b2 * Register_exp_avg_sq + (1.0f - b2) *
                                                         Register_param_grad *
                                                         Register_param_grad;
    float step = -lr * Register_exp_avg / (sqrt(Register_exp_avg_sq) + eps);

    param[p_idx] += step;
    exp_avg[p_idx] = Register_exp_avg;
    exp_avg_sq[p_idx] = Register_exp_avg_sq;
  }
}

void ADAM::adamUpdate(float* param,
                      const float* param_grad,
                      float* exp_avg,
                      float* exp_avg_sq,
                      const bool* tiles_touched,
                      const float lr,
                      const float b1,
                      const float b2,
                      const float eps,
                      const uint32_t N,
                      const uint32_t M) {
  const uint32_t cnt = N * M;
  adamUpdateCUDA<<<(cnt + 255) / 256, 256>>>(param, param_grad, exp_avg,
                                             exp_avg_sq, tiles_touched, lr, b1,
                                             b2, eps, N, M);
}
